#include "hip/hip_runtime.h"

#pragma once
#pragma warning(push, 0)
#include <stdio.h>
#include <stdlib.h>
#include <iostream>  
#include <math.h>
#include <thrust\device_ptr.h>
#include <thrust\sequence.h>
#include <>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#pragma warning(pop)
#include "parallelGpuNode2.cuh"
#include "Node.h"
#include "RandomGame.h"

#include "CUDA_def.h"


__global__ void PlayRandomGameGPU2(int n, int* results, char* boards, short boardSize, char* playersToMoveNext, char player)
{
	int i = blockIdx.x * blockDim.x * blockDim.y * blockDim.z + blockDim.x * blockDim.y * threadIdx.z + blockDim.x * threadIdx.y + threadIdx.x;
	if (i >= n) return;

	char* cBoard = boards + i * boardSize * boardSize / 2 * sizeof(char);
	char playerToMoveNext = playersToMoveNext[i];
	//SHARED MEMORY
	extern  __shared__ char s[];
	size_t offset = (blockDim.x * blockDim.y * threadIdx.z + blockDim.x * threadIdx.y + threadIdx.x);
	char* sharedBoard = s + offset;
	for (size_t i = 0; i < boardSize * boardSize / 2; i++) // sets single board in one shared memory bank to minimalize confilcts
	{
		sharedBoard[i * BANKSCOUNT] = cBoard[i];
	}
	Board board = Board(sharedBoard, boardSize);
	//~~SHARED MEMORY



	hiprandState_t state;
	hiprand_init(0, i, 1, &state);
	char result;

	PlayRandomGame(&board, playerToMoveNext, &result, [&]__device__(int minIn, int maxEx) {
		return hiprand(&state) % maxEx + minIn;
	});

	results[i] = GetResultPoints(result, player);

}

void CallPlayRandomGameGPU2(dim3 numBlocks, dim3 threadsPerBlock, int n, int* results, char* boards, short boardSize, char* playersToMoveNext, char player)
{
	size_t sharedMemorySize = threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z * boardSize * boardSize / 2 * sizeof(char);
	PlayRandomGameGPU2 << <numBlocks, threadsPerBlock, sharedMemorySize >> > (n, results, boards, boardSize, playersToMoveNext, player);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

}

